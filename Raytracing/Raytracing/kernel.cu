﻿#include "hip/hip_runtime.h"


#include <iostream>

#define K_TPB	256
#define K_ELEMS 25610

__global__ void sumaVectores(float *a, float *b, float *c, int total_elems) {
	int idx_ = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx_ < total_elems) {
		c[idx_] = a[idx_] + b[idx_];
	}
}

__global__ void sumaVectoresSuprema(float *a, float *b, float *c, int total_elems, int bLength) {
	int idx_ = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = 0; i < total_elems; i += bLength) {
		c[idx_] = a[idx_] + b[idx_];
	}
}

// Se ejecuta en la GPU y se llama desde la GPU
// __device__

// Se ejecuta en el host y se llama desde el host
// __host__

// SABER LIMITACION DE TARJETA GRAFICA -> device query (de CUDA)
int main(void) {
	float r, g, b;
	int nx = 200;
	int ny = 100;
	int ir, ig, ib;

	std::cout << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			r = float(i) / float(nx);
			g = float(j) / float(ny);
			b = 0.2;
			ir = int(255.99*r);
			ig = int(255.99*g);
			ib = int(255.99*b);
			std::cout << ir << " " << ig << " " << ib << "\n";
		}
	}

	// La finale
	getchar();
	exit(-1);







	// const int kNumElems = K_ELEMS;
	// const int knumBytes = sizeof(float) * kNumElems;
	// 
	// // Reservar la tarjeta grafica a la hora de realizar computo
	// cudaSetDevice(0);
	// 
	// float *h_a_ = (float*)malloc(knumBytes);
	// float *h_b_ = (float*)malloc(knumBytes);
	// float *h_c_ = (float*)malloc(knumBytes);
	// 
	// // if(h_a_ == NULL || h_b_ == NULL || h_c_ == NULL){
	// if (!h_a_ || !h_b_ || !h_c_) {
	// 	std::cerr << "Error al reservar memoria\n";
	// 	getchar();
	// 	exit(-1);
	// }
	// 
	// // Albert pone ++i
	// for (int i = 0; i < kNumElems; i++) {
	// 	h_a_[i] = rand() / (float)RAND_MAX;
	// 	h_b_[i] = rand() / (float)RAND_MAX;
	// }
	// 
	// float *d_a_ = NULL;
	// float *d_b_ = NULL;
	// float *d_c_ = NULL;
	// 
	// cudaMalloc((void**)&d_a_, knumBytes);
	// cudaMalloc((void**)&d_b_, knumBytes);
	// cudaMalloc((void**)&d_c_, knumBytes);
	// 
	// cudaMemcpy(d_a_, h_a_, knumBytes, cudaMemcpyHostToDevice);
	// cudaMemcpy(d_b_, h_b_, knumBytes, cudaMemcpyHostToDevice);
	// 
	// // Organizar bloques e hilos
	// int threads_per_block_ = K_TPB;
	// 
	// float tpb_aux = kNumElems / (float)(threads_per_block_);
	// int blocks_per_grid_ = kNumElems / threads_per_block_;
	// if (tpb_aux > blocks_per_grid_) {
	// 	blocks_per_grid_ += 1;
	// }
	// 
	// // Forma de cada thread
	// dim3 tpb_(threads_per_block_, 1, 1);
	// 
	// // Forma de la malla
	// dim3 bpg_(blocks_per_grid_, 1, 1);
	// 
	// // ----------------------------------------------
	// // Llamada a la funcion con DATOS QUE RESIDEN EN LA GRAFICA
	// // Si da el pequenyo error sint�ctico del <, no pasa nada
	// // sumaVectores<<< bpg_, tpb_ >>>(d_a_,d_b_,d_c_,kNumElems);
	// sumaVectoresSuprema << < bpg_, tpb_ >> > (d_a_, d_b_, d_c_, kNumElems, blocks_per_grid_*threads_per_block_);
	// // ----------------------------------------------
	// 
	// cudaError_t err_ = cudaGetLastError();
	// if (err_ != cudaSuccess) {
	// 	std::cerr << "Error " << cudaGetErrorString(err_) << "\n";
	// }
	// 
	// cudaMemcpy(h_c_, d_c_, knumBytes, cudaMemcpyDeviceToHost);
	// 
	// // Mirar el error
	// for (int i = 0; i < kNumElems; i++) {
	// 	if (fabs(h_a_[i] + h_b_[i] - h_c_[i]) > 1e-5) {
	// 		std::cerr << "Error en la posicion " << i << "\n";
	// 		getchar();
	// 		exit(-1);
	// 	}
	// }
	// 
	// // Liberacion de memoria
	// free(h_a_);
	// free(h_b_);
	// free(h_c_);
	// 
	// cudaFree(h_a_);
	// cudaFree(h_b_);
	// cudaFree(h_c_);
	// 
	// // Elimita todo lo de la grafica
	// cudaDeviceReset();
	// std::cout << "Optimo\n";
	// getchar();
	// exit(-1);

}
