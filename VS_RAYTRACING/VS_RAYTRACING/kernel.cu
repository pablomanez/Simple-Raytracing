#include "hip/hip_runtime.h"
// Librerias
#include "Util.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "hitable.h"
#include "camera.h"
#include "material.h"
#include "metal.h"
#include "lambertian.h"

__device__ glm::vec3 getColor(const ray& r, hitable **WORLD, int depth) {
	ray cur_ray = r;
	glm::vec3 cur_attenuation(1.0, 1.0, 1.0);
	
	for (int i = 0; i < depth; i++) {
		hit_record rec;
		if ((*WORLD)->hit(cur_ray, 0.001, FLT_MAX, rec)) {
			ray scattered;
			glm::vec3 attenuation;
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered)) {
				cur_attenuation *= attenuation;
				cur_ray = scattered;
			}
			else {
				return glm::vec3(0.0, 0.0, 0.0);
			}
		}
		else {
			glm::vec3 unit_direction = glm::normalize(cur_ray.getDirection());
			float t = 0.5f*(unit_direction.y + 1.0f);
			glm::vec3 c = (1.0f - t)*glm::vec3(1.0, 1.0, 1.0) + t * glm::vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return glm::vec3(0.0, 0.0, 0.0); // exceeded recursion

	/*
	hit_record rec;
	if (WORLD->hit(r, 0.001, FLT_MAX, rec)) {
		ray scattered;
		glm::vec3 attenuation;
		if (depth < 50 && rec.mat_ptr->scatter(r, rec, attenuation, scattered)) {
			return attenuation * getColor(scattered, WORLD, depth + 1);
		}
		else {
			return glm::vec3(0, 0, 0);
		}
	}
	else {
		glm::vec3 unitDirection = glm::normalize(r.getDirection());
		float t = 0.5 * (unitDirection.y + 1.0f);
		return ((1.0f - t) * glm::vec3(1.f)) + (t * glm::vec3(0.5, 0.7, 1.0));
	}
	*/
}

__global__ void initVariables(hitable **list, int list_length, hitable **WORLD) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		list[0] = new sphere(glm::vec3( 0,	     0,	-1),		0.5,		new lambertian	(glm::vec3(0.8, 0.3, 0.3)));
		list[1] = new sphere(glm::vec3( 0,  -100.5, -1),		100,		new lambertian	(glm::vec3(0.8, 0.8, 0.0)));
		list[2] = new sphere(glm::vec3( 1,	     0,	-1),		0.5,		new metal		(glm::vec3(0.8, 0.6, 0.2),	0.3));
		list[3] = new sphere(glm::vec3(-1,	     0,	-1),		0.5,		new metal		(glm::vec3(0.8, 0.8, 0.8),	1.0));
		*WORLD = new hitable_list(list, list_length);
	}
}

__global__ void createImage(glm::vec3 *d_arr, hitable **WORLD, int _w, int _h, int ns) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= _w || j >= _h) return;
	
	glm::vec3 color(0,0,0);
	camera cam;
	ray r;

	for (int s = 0; s < ns; s++) {
		float u = float(i + UTIL_rand_d()) / float(_w);
		float v = float(j + UTIL_rand_d()) / float(_h);
		r = cam.getRay(u, v);
		color += getColor(r, WORLD, 50);
	}

	color /= float(ns);
	// color = glm::vec3(glm::sqrt(color.r), glm::sqrt(color.g), glm::sqrt(color.b));

	color.r = int(255.99 * color.r);
	color.g = int(255.99 * color.g);
	color.b = int(255.99 * color.b);
	
	// Llevar los resultados al array 
	int pos = j * _w + i;
	pos = abs(_w*_h - pos);
	// d_arr[pos] = glm::vec3(pos, pos, pos);
	d_arr[pos] = color;
}

int main(void) {
	//Establezco la GPU que voy a usar
	hipSetDevice(0);
	
	// DATA 
	// ns es la precision del aliasing 
	const std::string filename = "img.PPM";
	int const TOTAL_HITABLES = 4;
	int _w, _h, ns;
	hipError_t err_;

	std::ofstream _out(filename);
	_w = 200;
	_h = 100;
	ns = 100;
	int imgSize = _w * _h;
	
	// Definir variables que voy a usar en la GPU
	// Array de salida de cada color
	glm::vec3 *d_arr = NULL;
	glm::vec3 *h_arr = new glm::vec3[_w*_h];

	for (int i = 0; i < imgSize; i++) {
		h_arr[i] = glm::vec3(0,0,0);
	}

	hitable **list = NULL;
	hitable **WORLD = NULL;

	// Reservo memoria de la GPU
	int kMemBytes = sizeof(glm::vec3)*_w*_h;
	
	hipMalloc((void**)&d_arr,	kMemBytes);
	hipMalloc((void**)&list,	TOTAL_HITABLES*sizeof(hitable*));
	hipMalloc((void**)&WORLD,  sizeof(hitable*));
	
	// Solo se lanza 1 nucleo ya que solo se va a hacer 1 vez
	initVariables <<< 1,1 >>> (list, TOTAL_HITABLES,WORLD);
	hipDeviceSynchronize();

	// Miro errores
	err_ = hipGetLastError();
	if (err_ != hipSuccess) {
		std::cerr << "A: Error " << hipGetErrorString(err_) << "\n";
	}

	// Defino el tamanyo de bloque y malla
	int K_THREADS_W	= 32;
	int K_THREADS_H	= 32;
	int K_BLOCKS_W	= ((float)_w / K_THREADS_W) > (_w / K_THREADS_W) ? (_w / K_THREADS_W) + 1 : (_w / K_THREADS_W);
	int K_BLOCKS_H	= ((float)_h / K_THREADS_H) > (_h / K_THREADS_H) ? (_h / K_THREADS_H) + 1 : (_h / K_THREADS_H);

	std::cout << "BLOQUE: " << K_THREADS_W << " x " << K_THREADS_H << std::endl;
	std::cout << "MALLA: " << K_BLOCKS_W << " x " << K_BLOCKS_H << std::endl;

	dim3 tpb(K_THREADS_W, K_THREADS_H, 1);	// Hilos por bloque		(THREADS PER BLOCK)
	dim3 bpg(K_BLOCKS_W, K_BLOCKS_H, 1);	// Bloques por malla	(BLOCKS PER GRID)
	
	// LINK STARTO!
	hipMemcpy(d_arr, h_arr, kMemBytes, hipMemcpyHostToDevice);
	// Miro errores
	err_ = hipGetLastError();
	if (err_ != hipSuccess) {
		std::cerr << "C: Error " << hipGetErrorString(err_) << "\n";
	}

	createImage <<< bpg, tpb >>> (d_arr, WORLD, _w, _h, ns);
	// Miro errores
	err_ = hipGetLastError();
	if (err_ != hipSuccess) {
		std::cerr << "D: Error " << hipGetErrorString(err_) << "\n";
	}

	hipMemcpy(h_arr, d_arr, kMemBytes, hipMemcpyDeviceToHost);
	
	// Miro errores
	err_ = hipGetLastError();
	if (err_ != hipSuccess) {
		std::cerr << "E: Error " << hipGetErrorString(err_) << "\n";
	}

	// Inicio del archivo PPM
	_out << "P3\n" << _w << " " << _h << "\n255\n";
	
	// ESTO DEBE IR EN UNA FUNCION GLOBAL
	for (int i = 0; i < imgSize; i++) {
			_out << h_arr[i].r << " ";
			_out << h_arr[i].g << " ";
			_out << h_arr[i].b << "\n";
	}

	// Libero la GPU
	// TODO: 
	//		Liberar el espacio reservado de la gpu
	// delete h_arr;
	
	// hipFree(d_arr);
	// hipFree(list);
	// hipFree(WORLD);
	
	hipDeviceReset();

	std::cout << "Finalizado" << '\n';
}

// PRUEBA CON GLM QUE FUNCIONA
/*

__global__ void molona(glm::vec3 *s_ptr, glm::vec3 *_arr) {
	int idx_ = blockIdx.x * blockDim.x + threadIdx.x;
	
	float mult = idx_ + 1;
	s_ptr[idx_] = mult * _arr[idx_];
}

int main(void) {
	//Establezco la GPU que voy a usar
	hipSetDevice(0);

	////////////////////////////////////////////
	// DEFINIR ESTRUCTURA DE LOS BLOQUES
	#define K_THREADS	3
	#define K_BLOCKS	1

	glm::vec3 *d_arr = NULL;				// DEVICE
	glm::vec3 *h_arr = new glm::vec3[3];	// HOST
		h_arr[0] = glm::vec3(1, 1, 1);
		h_arr[1] = glm::vec3(1, 1, 1);
		h_arr[2] = glm::vec3(1, 1, 1);

	// Memoria a reservar para la GPU
	int mem = sizeof(glm::vec3) * 3;

	// Reserva de memoria en la CPU
	// NOTA: Creo que ambas formas son v�lidas
	glm::vec3 *d_str = NULL;				// DEVICE
	glm::vec3 *h_str = new glm::vec3[3];	// HOST
	// glm::vec3 *h_str = (glm::vec3*)malloc(mem);

	// Reservo memoria en la GPU
	hipMalloc((void**)&d_str, mem);
	hipMalloc((void**)&d_arr, mem);

	hipMemcpy(d_arr, h_arr, mem, hipMemcpyHostToDevice);

	dim3 tpb(K_THREADS, 1, 1);	// Hilos por bloque		(THREADS PER BLOCK)
	dim3 bpg(K_BLOCKS, 1, 1);	// Bloques por malla	(BLOCKS PER GRID)

	molona << < bpg, tpb >> > (d_str, d_arr);

	// hipError_t err_ = hipGetLastError();
	// if (err_ != hipSuccess) {
	// 	std::cerr << "Error " << hipGetErrorString(err_) << "\n";
	// }

	hipMemcpy(h_str, d_str, mem, hipMemcpyDeviceToHost);
	hipMemcpy(h_arr, d_arr, mem, hipMemcpyDeviceToHost);

	for (int i = 0; i < 3; i++) {
		std::cout << "ORIGINAL: (" << h_arr[i].x << "," << h_arr[i].y << "," << h_arr[i].z << ")" << std::endl;
	}
	for (int i = 0; i < 3; i++) {
		std::cout << "    HOST: (" << h_str[i].x << "," << h_str[i].y << "," << h_str[i].z << ")" << std::endl;
	}
	
	//Libero la GPU
	hipDeviceReset();

	std::cout << "Finalizado" << '\n';
}
*/

// Cuanta memoria tengo que reservar en la GPU?
// DATA			->	SIZE (bytes)
// -------------  -------------- ----------------------------------- ----------- --- -
// camera		->	48			: 4*glm::vec3						: 4*12		= 48
// sphere		->	32			: glm::vec3+float+material+hitable	: 12+4+8+8	= 32
// lambertian	->	24			: glm::vec3+material+?????			: 12+8+4	= 24
// metal		->	24			: glm::vec3+float+material			: 12+4+8	= 24
// hitable_list	->	24			: hitable**+int+hitable+?????		: 8+2+8+6	= 24
// ray			->	24			: 2*glm::vec3						: 2*12		= 24
// glm::vec3	->	12			: 3*float							: 3*4		= 12
// material		->	8
// hitable		->	8
// double		->	8
// float		->	4
// int/bool		->	2
// --------------------------------------------------------------------------------- -

/*
NOTAS:
	Cada hilo ataca a un pixel de la imagen y se encarga de computar un color
	Ese color se almacena en la misma posicion que se le ha dado al del hilo en el total
	Es decir:
		(EJEMPLO: HILOS/BLOQUE = 100)
		El HILO 0 del BLOQUE 0 escribira en la posicion 0 del array	de salida
		El HILO 50 del BLOQUE 0 escribira en la posicion 50 del array de salida
		El HILO 10 del BLOQUE 1 escribira en la posicion 110 del array de salida
		El HILO 99 del BLOQUE 2 escribira en la posicion 299 del array de salida

	Despu�s, ese array DEBE SER llevado, usando la CPU, al fichero de salida para tener la imagen final

OBSERVACIONES:
	Un hilo utiliza los mismos datos para computar el color que cualquier otro hilo que se lance -> MEMORIA COMPARTIDA?

COMO COJONES LO HE HECHO:
	Cambiar la estructura de las clases, porque no compila ^^
	Mover todo al .h
	Leer MUCHA documentaci�n
	Nvidia GTX 660 -> 16 bloques/sm && 2048 hilos/sm (IMG DE 200x100!!!!)
		T_BLOQUE = 32x32
		T_MALLA = 200/32 x 100/32 = 6.25x3.125 = 7x4

	---
	Hay un error y como da en la primera posicion de todas, sale de la funcion directamente y no hace nada

PASOS:
	1. Reservar la GPU y la memoria a usar
	2. Organizar bloques e hilos
	3. Llamada a funcion/es
	4. Liberacion de la memoria y la GPU
*/
// https://devblogs.nvidia.com/accelerated-ray-tracing-cuda/
// --------------------------------------------