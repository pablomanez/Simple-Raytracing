#include "hip/hip_runtime.h"
#include ""

// Librerias
#include <iostream>
#include <fstream>

#define GLM_FORCE_PURE
#include <glm/glm.hpp>

int main(void) {
	glm::vec3 color;
	int ir, ig, ib;
	int nx = 200;	// Ancho
	int ny = 100;	// Alto

	std::ofstream _out("img.PPM");

	// Inicio del archivo PPM
	//	P3  -> El archivo esta en ASCII
	//	255 -> Color 'maximo'
	_out << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			color = glm::vec3(
				float(i) / float(nx),
				float(j) / float(ny),
				0.2
			);

			ir = int(255.99 * color.r);
			ig = int(255.99 * color.g);
			ib = int(255.99 * color.b);

			// Valores de los pixeles
			_out << ir << " " << ig << " " << ib << "\n";
		}
		std::cout << j << '\n';
	}
	std::cout << "Finalizado" << '\n';

	// La finale
	getchar();
	exit(-1);


}
