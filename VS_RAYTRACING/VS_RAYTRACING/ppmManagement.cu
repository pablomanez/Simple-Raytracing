#include "hip/hip_runtime.h"
#include "ppmManagement.h"

ppmManagement::ppmManagement(){}
ppmManagement::~ppmManagement(){}

glm::vec3 ppmManagement::getColor(const ray& r, hitable *WORLD, int depth) {
	hit_record rec;
	if (WORLD->hit(r, 0.001, FLT_MAX, rec)) {
		ray scattered;
		glm::vec3 attenuation;
		if (depth < 50 && rec.mat_ptr->scatter(r,rec,attenuation,scattered)) {
			return attenuation * getColor(scattered, WORLD, depth + 1);
		}
		else {
			return glm::vec3(0, 0, 0);
		}
	}
	else {
		glm::vec3 unitDirection = glm::normalize(r.getDirection());
		float t = 0.5 * (unitDirection.y + 1.0f);
		return ((1.0f - t) * glm::vec3(1.f)) + (t * glm::vec3(0.5, 0.7, 1.0));
	}
}

void ppmManagement::createImage(int _w, int _h, int ns, const std::string &_name) {
	// ns es la precision del aliasing 
	// ns = 100 (default)
	std::ofstream _out(_name);
	int ir, ig, ib;
	float u,v;
	ray r;
	glm::vec3 color;

	int const total_hitables = 4;
	hitable *list[total_hitables];
	list[0] = new sphere(glm::vec3(0, 0, -1), 0.5, new lambertian(glm::vec3(0.8, 0.3, 0.3)));
	list[1] = new sphere(glm::vec3(0, -100.5, -1), 100, new lambertian(glm::vec3(0.8, 0.8, 0.0)));
	list[2] = new sphere(glm::vec3(1,0,-1), 0.5, new metal(glm::vec3(0.8, 0.6, 0.2),0.3));
	list[3] = new sphere(glm::vec3(-1,0,-1), 0.5, new metal(glm::vec3(0.8, 0.8, 0.8),1.0));
	hitable *WORLD = new hitable_list(list, total_hitables);

	camera cam;
	
	// Inicio del archivo PPM
	//	P3  -> El archivo esta en ASCII
	//	255 -> Color 'maximo'
	_out << "P3\n" << _w << " " << _h << "\n255\n";
	for (int j = _h - 1; j >= 0; j--) {
		for (int i = 0; i < _w; i++) {
			color = glm::vec3(0, 0, 0);
			for (int s = 0; s < ns; s++) {
				u = float(i + UTIL_rand_d()) / float(_w);
				v = float(j + UTIL_rand_d()) / float(_h);
				r = cam.getRay(u, v);
				color += getColor(r, WORLD,0);
			}

			color /= float(ns);
			color = glm::vec3(glm::sqrt(color.r), glm::sqrt(color.g), glm::sqrt(color.b));
			ir = int(255.99 * color.r);
			ig = int(255.99 * color.g);
			ib = int(255.99 * color.b);

			// Valores de los pixeles
			_out << ir << " ";
			_out << ig << " ";
			_out << ib << "\n";
		}
	}
}
