#include "hip/hip_runtime.h"
#include "ppmManagement.h"

ppmManagement::ppmManagement(){}
ppmManagement::~ppmManagement(){}

glm::vec3 ppmManagement::getColor(const ray& r, hitable *WORLD) {
	hit_record rec;
	if (WORLD->hit(r, 0.001, FLT_MAX, rec)) {
		glm::vec3 target = rec.p + rec.normal + randomInUnitSphere();
		ray ray_ret(rec.p, target - rec.p);
		return 0.5f*getColor(ray_ret,WORLD);
	}
	else {
		glm::vec3 unitDirection = glm::normalize(r.getDirection());
		float t = 0.5 * (unitDirection.y + 1.0f);
		return ((1.0f - t) * glm::vec3(1.f)) + (t * glm::vec3(0.5, 0.7, 1.0));
	}
}

void ppmManagement::createImage(int _w, int _h, int ns, const std::string &_name) {
	// ns es la precision del aliasing 
	// ns = 100 (default)
	std::ofstream _out(_name);
	int ir, ig, ib;
	float u,v;
	ray r;
	glm::vec3 color;

	int const total_hitables = 2;
	hitable *list[total_hitables];
	list[0] = new sphere(glm::vec3(0, 0, -1), 0.5);
	list[1] = new sphere(glm::vec3(0, -100.5, -1), 100);
	hitable *WORLD = new hitable_list(list, total_hitables);

	camera cam;
	
	// Inicio del archivo PPM
	//	P3  -> El archivo esta en ASCII
	//	255 -> Color 'maximo'
	_out << "P3\n" << _w << " " << _h << "\n255\n";
	for (int j = _h - 1; j >= 0; j--) {
		for (int i = 0; i < _w; i++) {
			color = glm::vec3(0, 0, 0);
			for (int s = 0; s < ns; s++) {
				u = float(i + UTIL_rand_d()) / float(_w);
				v = float(j + UTIL_rand_d()) / float(_h);
				r = cam.getRay(u, v);
				color += getColor(r, WORLD);
			}

			color /= float(ns);
			color = glm::vec3(glm::sqrt(color.r), glm::sqrt(color.g), glm::sqrt(color.b));
			ir = int(255.99 * color.r);
			ig = int(255.99 * color.g);
			ib = int(255.99 * color.b);

			// Valores de los pixeles
			_out << ir << " ";
			_out << ig << " ";
			_out << ib << "\n";
		}
	}
}
