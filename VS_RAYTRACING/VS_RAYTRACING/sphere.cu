#include "hip/hip_runtime.h"
#include "sphere.h"

sphere::sphere() {}
sphere::~sphere() {}

sphere::sphere(glm::vec3 cen, float r): center(cen), radius(r){}

bool sphere::hit(const ray &r, float t_min, float t_max, hit_record &rec) const{
	glm::vec3 const origin = r.getOrigin();
	glm::vec3 const dir = r.getDirection();

	glm::vec3 oc = origin - center;
	float a = glm::dot(dir, dir);
	float b = glm::dot(oc, dir);
	float c = glm::dot(oc, oc) - radius * radius;
	float disc = b*b - a*c;

	if (disc > 0) {
		float temp = (-b - glm::sqrt(b*b - a*c)) / a;
		if (temp < t_max && temp > t_min) {
			rec.t = temp;
			rec.p = r.getPointAtParameter(rec.t);
			rec.normal = (rec.p - center) / radius;
			return true;
		}

		temp = (-b + glm::sqrt(b*b - a*c)) / a;
		if (temp < t_max && temp > t_min) {
			rec.t = temp;
			rec.p = r.getPointAtParameter(rec.t);
			rec.normal = (rec.p - center) / radius;
			return true;
		}
	}
	return false;
}
